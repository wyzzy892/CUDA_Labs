#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cmath>


#define M 20
#define N 20
#define BLOCK_SIZE 20
#define BASE_TYPE int 


using namespace std;

// kernel
__global__ void Add(const BASE_TYPE* a, const BASE_TYPE* b, BASE_TYPE* c)
{
	int idx = N * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	c[idx] = a[idx] + b[idx];
}


int main()
{
	// переменные на CPU
	BASE_TYPE a[M][N] = { 0 };
	BASE_TYPE b[M][N] = { 0 };
	BASE_TYPE c[M][N] = { 0 };

	// инициализация rand для float
	srand(time(0));
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < N; j++)
		{
			a[i][j] = (BASE_TYPE)rand() %100+14;
			b[i][j] = (BASE_TYPE)rand() %100+23;
		}
	}

	BASE_TYPE* dev_a = NULL, * dev_b = NULL, *dev_c = NULL;

	//выделение памяти на GPU
	size_t size = N * M * sizeof(BASE_TYPE);
	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	// копирование информации с CPU на GPU
	hipMemcpy(dev_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, size, hipMemcpyHostToDevice);


	dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid = dim3(N / BLOCK_SIZE, M / BLOCK_SIZE);

	// вызов ядра
	Add << <blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, dev_c);

	// копирование результата работы ядра с GPU на CPU
	hipMemcpy(&c, dev_c, size, hipMemcpyDeviceToHost);

	// вывод информации
	for (int i = 0; i < M; i++)
	{
		for (int j = 0; j < N; j++)
			cout << c[i][j] << ' ';
		cout << endl;
	}
	cout << endl;

	// очищаем память на GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}