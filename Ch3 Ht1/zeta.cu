#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

#define N 100

using namespace std;

// считаем y = sqrt(1-x^2)
__global__ void dzeta(double* zeta, double *s) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	zeta[index] = 1.0 / powf((index + 1), *s);
}


int main()
{
	// переменные на CPU
	double zeta[N]; //zeta
	double s=5; //s - степень
	double sum = 0; // для вычисления суммы

	// переменные на GPU
	double* dev_zeta, * dev_s;

	// выделяем память на GPU
	hipMalloc((void**)&dev_zeta, N*sizeof(double));
	hipMalloc((void**)&dev_s, sizeof(double));

	// копирование информации с CPU на GPU
	hipMemcpy(dev_s, &s, sizeof(double), hipMemcpyHostToDevice);


	// вызов ядра
	dzeta << < 1, N >> > (dev_zeta, dev_s);

	// копирование результата работы ядра с GPU на CPU
	hipMemcpy(&zeta, dev_zeta, N*sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < N; ++i)
		sum += zeta[i];

	// вывод информации
	cout << "Zeta functions is: " << sum << endl;
	// очищение памяти на GPU
	hipFree(dev_zeta);
	hipFree(dev_s);

	return 0;
}