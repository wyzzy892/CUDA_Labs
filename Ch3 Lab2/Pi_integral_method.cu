#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

#define N 1000

using namespace std;

// считаем y = sqrt(1-x^2)
__global__ void f(double* mas) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	double x = double(index) / double(N);
	mas[index] = sqrt(1 - x*x);
}


int main()
{
	double mas[N];
	double result=0;
	double* dev_mas;
	hipMalloc((void**)&dev_mas, N*sizeof(double));

	//вызов ядра
	f << <1, N >> > (dev_mas);

	hipMemcpy(&mas, dev_mas, N*sizeof(double), hipMemcpyDeviceToHost);
	//проверка на ошибку
	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("%s ", hipGetErrorString(err));

	// копирование результата работы ядра с GPU на CPU
	hipMemcpy(&mas, dev_mas, N * sizeof(double), hipMemcpyDeviceToHost);
	//Считаем площадь
	for (int i = 0; i < N; i++) {
		result += 2*mas[i];
	}
	// вывод результата
	cout<<"Pi = "<<4.0 * result / (2*N)<<endl;
	// очищение памяти на GPU
	hipFree(dev_mas);
	return 0;
}