#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>

#define N 360
#define M 500

using namespace std;

// Функция инициализации начальных состояний генератора случайных чисел для каждого потока
__global__ void init(hiprandState* state) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(1234, index, 0, &state[index]);
}

//Функция генерерующая случайное число из равномерного распределения 
__device__ double generate(hiprandState* state, int index) {
	hiprandState local_state = state[index];
	double random_value = hiprand_uniform_double(&local_state);
	state[index] = local_state;
	return random_value;
}

__global__ void MonteCarlo(hiprandState* state, int* n, int* n_circle) {
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < N * M) {
		double x = generate(state, index);
		double y = generate(state, index);
		if (x * x + y * y <= 1)
			atomicAdd(n_circle, 1);
		atomicAdd(n, 1);
	}
}

int main()
{
	setlocale(LC_ALL, "rus");
	hiprandState* states;
	hipMalloc((void**)&states, N * M * sizeof(hiprandState));
	int n = 0, n_circle = 0;
	int* dev_n, * dev_n_circle;
	hipMalloc((void**)&dev_n, sizeof(int));
	hipMalloc((void**)&dev_n_circle, sizeof(int));
	hipMemcpy(dev_n, &n, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_n_circle, &n_circle, sizeof(int), hipMemcpyHostToDevice);

	// замеряем время
	float gpu_elapsed_time;
	hipEvent_t gpu_start, gpu_stop;
	hipEventCreate(&gpu_start);
	hipEventCreate(&gpu_stop);
	hipEventRecord(gpu_start, 0);

	//инициализируем состояния
	init << <N, M >> > (states);

	//считаем точки
	MonteCarlo << <N, M >> > (states, dev_n, dev_n_circle);
	hipMemcpy(&n, dev_n, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&n_circle, dev_n_circle, sizeof(int), hipMemcpyDeviceToHost);

	// замеряем время
	hipEventRecord(gpu_stop, 0);
	hipEventSynchronize(gpu_stop);
	hipEventElapsedTime(&gpu_elapsed_time, gpu_start, gpu_stop);
	hipEventDestroy(gpu_start);
	hipEventDestroy(gpu_stop);

	// считаем число pi
	cout << "Число пи приближенно равно " << 4.0 * n_circle / n<<endl<<gpu_elapsed_time/1000<<"";

	//Освобождаем память
	hipFree(dev_n);
	hipFree(dev_n_circle);
	return 0;
}