#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <cmath>

#define N 50

using namespace std;

__global__ void DotProduct(int* a, int* b, int* res) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        int component = a[index] * b[index];
        atomicAdd(res, component);
    }
}


int main()
{
    int a[N], b[N], res;  // объявляем массивы для CPU
    int* dev_a, * dev_b, * dev_res;  // объявляем массивы для GPU

    // инициализируем массивы a, b значениями
    for (int i = 0; i < N; i++) {
        a[i] = pow(i + 1, 2);
        b[i] = pow(i + 1, 2);
    }
    res = 0;

    //выделяем память на GPU
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_res, sizeof(int));

    //копируем данные из CPU на GPU
    hipMemcpy(dev_a, &a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, &b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_res, &res, sizeof(int), hipMemcpyHostToDevice);

    //вызываем функцию ядра с 5 блоками по 10 нитей
    DotProduct << <5, 10 >> > (dev_a, dev_b, dev_res);

    //копируем данные обратно на хост
    hipMemcpy(&res, dev_res, sizeof(int), hipMemcpyDeviceToHost);

    //Вывод значения
    cout << res<<endl;

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_res);

    return 0;
}
